#include "hip/hip_runtime.h"
#include "matmul.h"

#include <cstdio>
#include <vector>

static float *weights { nullptr };
static std::vector<float *> deviceMemory;

#define HANDLE_CUDA_RESULT(FUNC) \
    do { \
        if (hipError_t result = FUNC; result != hipSuccess) \
        { \
            fprintf(stderr, "Encountered cuda error with function '%s' at line %d: %s(%d)\n", #FUNC, __LINE__, hipGetErrorName(result), result); \
            exit(1); \
        } \
    } while(0)

static bool isInDeviceMemory(float *ptr)
{
    for(const auto p : deviceMemory)
    {
        if (p == ptr)
        {
            return true;
        }
    }

    return false;
}

float *allocateDeviceWeights(void *data, size_t size)
{
    if (weights)
    {
        HANDLE_CUDA_RESULT(hipFree(weights));
    }

    HANDLE_CUDA_RESULT(hipMalloc((void**)&weights, size));
    HANDLE_CUDA_RESULT(hipMemcpy(weights, data, size, hipMemcpyHostToDevice));

    return weights;
}

float *allocateDeviceMemory(float *source, size_t size)
{
    float *ptr{nullptr};
    HANDLE_CUDA_RESULT(hipMalloc((void**)&ptr, size));
    HANDLE_CUDA_RESULT(hipMemcpy(ptr, source, size, hipMemcpyHostToDevice));
    deviceMemory.push_back(ptr);
    return ptr;
}

float *allocatePinnedHostMemory(size_t size)
{
    float *ptr{nullptr};
    HANDLE_CUDA_RESULT(hipHostMalloc((void**)&ptr, size));
    deviceMemory.push_back(ptr);
    return ptr;
}

void freeDeviceMemoryAndWeights()
{
    if (weights)
    {
        HANDLE_CUDA_RESULT(hipFree(weights));
        weights = nullptr;
    }

    for (auto ptr : deviceMemory)
    {
        HANDLE_CUDA_RESULT(hipFree(ptr));
    }
    deviceMemory.clear();
}

__global__ void matrixMultiplicationKernel(float* w, float* x, float* out, int n, int d) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum {0.0f};

    if (col < d)
    {
        for (int i = 0; i < n; i++) {
            //printf("COL: %d i: %d w: %f x: %f\n", col, i, w[col*n+i], x[i]);
            sum += w[col * n + i] * x[i];
        }
    }

    //printf("COL: %d n: %d d: %d sum: %f\n", col, n, d, sum);
    out[col] = sum;
}

// W (d,n) @ x (n,) -> xout (d,)
void matmul(float *h_out, float *h_x, float *h_w, int n, int d) {
    static bool isCudaChecked {false};

    if (isCudaChecked == false) {
        int deviceCnt;
        HANDLE_CUDA_RESULT(hipGetDeviceCount(&deviceCnt));

        if (deviceCnt < 1) {
            fprintf(stderr, "No CUDA devices found.\n");
            exit(1);
        }

        isCudaChecked = true;
    }

    const size_t size_w = sizeof(float) * (n * d);
    const size_t size_x = sizeof(float) * (n);
    const size_t size_out = sizeof(float) * (d);

    float *d_x{};
    float *d_w{};
    float *d_out{};

    // Allocate device memory
    HANDLE_CUDA_RESULT(hipMalloc((void **) &d_w, size_w));
    HANDLE_CUDA_RESULT(hipMalloc((void **) &d_x, size_x));
    HANDLE_CUDA_RESULT(hipMalloc((void **) &d_out, size_out));

    if (isInDeviceMemory(h_w) == false)
        HANDLE_CUDA_RESULT(hipMemcpy(d_w, h_w, size_w, hipMemcpyHostToDevice));
    else
        d_w = h_w;

    if (isInDeviceMemory(h_x) == false)
        HANDLE_CUDA_RESULT(hipMemcpy(d_x, h_x, size_x, hipMemcpyHostToDevice));
    else
        d_x = h_x;

    dim3 threadsPerBlock{static_cast<unsigned>(d)};
    dim3 blocksPerGrid{1};
    if (d > 512) {
        threadsPerBlock.x = 512;
        blocksPerGrid.x = ceil(double(d) / double(threadsPerBlock.x));
    }

    matrixMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(d_w, d_x, d_out, n, d);
    HANDLE_CUDA_RESULT(hipDeviceSynchronize());


    HANDLE_CUDA_RESULT(hipMemcpy(h_out, d_out, size_out, hipMemcpyDeviceToHost));
    HANDLE_CUDA_RESULT(hipDeviceSynchronize());

    // Deallocate device memory
    if (d_x != h_x)
    {
        HANDLE_CUDA_RESULT(hipFree(d_x));
    }
    if (d_w != h_w)
    {
        HANDLE_CUDA_RESULT(hipFree(d_w));
    }
    HANDLE_CUDA_RESULT(hipFree(d_out));
}